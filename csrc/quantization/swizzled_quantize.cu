#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// SPDX-License-Identifier: Apache-2.0

// DeepSpeed Team
#include "../cppdebug.h"
#include "../cudebug.cuh"
#include "memory_access_utils.h"
#include "quantization_utils.h"
#include "reduction_utils.h"

using rop = reduce::ROpType;

namespace swiz_quant {
// swiz_quant命名空间内定义了一些常量，包括最大线程数、最小线程数、
// 步长粒度以及每步处理的元素数量。这些值都是在量化过程中使用的。
constexpr int max_threads = 512;
constexpr int min_threads = 32;

constexpr int step_granularity = 2;
constexpr int h_per_step = step_granularity * quantize::h_per_load;
}  // namespace swiz_quant


// swizzled_quant_kernel是一个模板函数，它的模板参数包括：量化位数numBits、
// 总块数totalChunks、线程数threads、以及量化类型quantType。
//它接受的参数包括量化后的数据、量化比例尺、未压缩的数据、每个分组的元素数、节点数、每个节点的设备数。
template <int numBits, int totalChunks, int threads, quantize::Type quantType>
__global__ void swizzled_quant_kernel(int8_t* quantized_data,
                                      float* quantized_scales,
                                      const __half* uncompressed_data,
                                      int elems_per_group,
                                      int nodes,
                                      int devices_per_node)
{
    debuginfo();

    // 获取当前的线程块对象（thread block）。hw_warp_size是一个常量32
    cg::thread_block tb = cg::this_thread_block();
    // 从线程块中划分一个大小为硬件warp大小的分区（warp）。
    cg::thread_block_tile<hw_warp_size> warp = cg::tiled_partition<hw_warp_size>(tb);

    // Indexing offsets, same as normal quantization for in-case
    // 计算线程块在网格中的全局排序（rank）。这里网格是3维的，每个维度可能包含多个线程块
    const int block_rank = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    // 根据线程块的全局排序和每组的元素数量来计算偏移量。
    const int block_offset = block_rank * elems_per_group;
    // quantize::h_per_load 的定义在 `DeepSpeed/csrc/includes/quantization_utils.h` 中的：
    // constexpr int granularity = 16;
    // constexpr int h_per_load = granularity / sizeof(__half);
    // 计算在一个线程块中的线程的偏移量。这里假设一个线程将加载quantize::h_per_load个元素。
    const int elem_offset = tb.thread_index().x * quantize::h_per_load;
    // 计算基础偏移量，即线程块偏移量和线程偏移量的和。
    const int base_offset = block_offset + elem_offset;
    // 计算步长。步长是一个线程块的大小乘以每个线程加载的元素数量。
    const int stride = tb.size() * quantize::h_per_load;
    // 根据基础偏移量获取未压缩数据的指针。
    const __half* input_base = uncompressed_data + base_offset;

    // Local buffer
    // 在本地声明一个缓冲区，用来存储加载的数据。这里__half2是CUDA中用于表示半精度浮点数的类型。
    __half2 local_buffer[totalChunks * quantize::h2_per_load];

    quantize::GroupStats<quantType> stats; // 声明一个GroupStats对象，用来存储统计信息。
#pragma unroll // 是一个编译指令，它告诉编译器展开接下来的循环，可以提高代码的执行效率。
    // 然后是一个循环，读取全局内存的数据并存储到本地缓冲区，然后更新统计信息。
    for (int i = 0; i < totalChunks; i++) {
        __half2* iteration_buffer = local_buffer + i * quantize::h2_per_load;

        mem_access::load_global<quantize::granularity>(
            iteration_buffer, input_base + i * stride, elem_offset + i * stride < elems_per_group);

#pragma unroll
        for (int j = 0; j < quantize::h2_per_load; j++) { stats.update(iteration_buffer[j]); }
    }

    // 调用get_params函数从统计对象（stats）中获取量化参数。这些参数包括每个矢量的缩放因子和零点。
    // 此行中numBits和threads是模板参数，分别表示量化的位数和线程数量。同时，tb和warp分别表示线程块和线程束的对象。
    auto params = stats.template get_params<numBits, threads>(tb, warp);

    // 设置partition_id为z方向的block索引。
    const int partition_id = blockIdx.z;
    // 计算每个节点的设备偏移，即当前分区ID除以每个节点的设备数。
    const int partition_offset = partition_id / devices_per_node;
    // 计算分区基数，即当前分区ID除以每个节点的设备数的余数乘以节点数。
    const int partition_base = (partition_id % devices_per_node) * nodes;
    // 计算流水线偏移，即y方向的block索引乘以设备总数。
    const int pipelining_offset = blockIdx.y * (devices_per_node * nodes);
    // 计算输出分区，即流水线偏移加上分区基数和设备偏移。
    const int output_partition = (pipelining_offset + partition_base + partition_offset);

    // 计算输出标量效应，即每个字节可以包含的元素数量。
    constexpr int out_scalar_effect = 8 / numBits;
    // 计算输出block的排名，即输出分区乘以x方向的grid大小加上x方向的block索引。
    const int out_block_rank = output_partition * gridDim.x + blockIdx.x;
    // 计算输出block的偏移，即输出block的排名乘以每个组的元素数除以输出标量效应。
    const int out_block_offset = out_block_rank * elems_per_group / out_scalar_effect;
    // 计算输出基础偏移，即输出block的偏移加上元素偏移除以输出标量效应。
    const int out_base_offset = out_block_offset + elem_offset / out_scalar_effect;
    // 计算输出基地址，即量化数据加上输出基础偏移。
    int8_t* out_base = quantized_data + out_base_offset;

    // 计算输出步长，即步长除以输出标量效应。
    const int out_stride = stride / out_scalar_effect;
    // 计算每次输出的int8数目，即每次加载的半精度浮点数数量除以输出标量效应。
    constexpr int num_int8_out = quantize::h_per_load / out_scalar_effect;

    // 如果当前线程是线程块中的第一个线程，那么将参数存储到指定的位置。
    if (tb.thread_index().x == 0) { params.store(quantized_scales, out_block_rank); }

#pragma unroll
    // 对每个块进行循环。
    for (int i = 0; i < totalChunks; i++) {
        // 如果当前元素在有效范围内，则执行以下操作：
        if (i * stride + elem_offset < elems_per_group) {
            // 定义一个本地输出数组，用于临时存储量化的结果。
            int8_t local_output[quantize::h_per_load / out_scalar_effect];
            // 进行量化操作，结果存储在local_output中。
            quantize::_chunk<numBits, quantType>(
                local_output, local_buffer + i * quantize::h2_per_load, params);
            // 将本地的量化结果存储到全局内存中。
            mem_access::store_global<num_int8_out>(out_base + i * out_stride, local_output);
        }
    }
}

#define LAUNCH_SWIZZLE_QUANT(total_chunks, threads)                                           \
    swizzled_quant_kernel<numBits, total_chunks, threads, qType><<<grid, block, 0, stream>>>( \
        q_data, q_scales, input_data, elems_per_group, nodes, devices_per_node);
// 这里解释了 "Swizzled quantization"（交错量化）是如何工作的。
// 这种方法主要是为了优化多节点多设备的并行计算中的通信效率。
// 这里给出了一个在两个节点，每个节点上有四个设备的情况下的划分示例。
// 原始的数据划分可能是线性的，比如0-7每个数代表一组数据，且数据在设备上的存储是连续的：
//  --- --- --- --- --- --- --- ---
// | 0 | 1 | 2 | 3 | 4 | 5 | 6 | 7 |
//  --- --- --- --- --- --- --- ---
// 在交错量化中，数据会被重新组织，变成如下形式：
//  --- --- --- --- --- --- --- ---
// | 0 | 4 | 1 | 5 | 2 | 6 | 3 | 7 |
// --- --- --- --- --- --- --- ---
// 此处，每个数字代表一组数据，你可以看到原本连续存储的数据被"交错"了。
// 在这个例子中，0和4可能在同一个节点的不同设备上，1和5在另一个节点的不同设备上。
// 通过这种方式，我们可以在进行节点间的通信时，同时从每个节点的多个设备中获取数据，这样可以提高通信效率。
// 还提到了一个"分片"的概念，比如说二分分片。在这种情况下，每个分区的前半部分数据会被连接在一起，
// 这样可以为后续的流水线操作提供更好的支持。


// 这段代码是一个模板函数，实现了"Swizzled quantization"的过程。
// 主要参数包括量化数据q_data，量化比例尺q_scales，输入数据input_data，分组数量groups，
// 每组元素数量elems_per_group，流水线大小pipelining，节点数nodes
// 和每个节点上的设备数devices_per_node。最后一个参数stream是用于CUDA的异步并行执行的流。
/*
Swizzled quantization reorganizes the quantized groups in order to better facilitate
communication. As an example of the partitioning scheme we have the following example
of 2 node, 4 device swizzling:

 --- --- --- --- --- --- --- ---
| 0 | 1 | 2 | 3 | 4 | 5 | 6 | 7 |
 --- --- --- --- --- --- --- ---
becomes
 --- --- --- --- --- --- --- ---
| 0 | 4 | 1 | 5 | 2 | 6 | 3 | 7 |
 --- --- --- --- --- --- --- ---

Multiple quantization groups may be mapped into a single partition. In order to better support
later pipelining, we may also perform an additional slicing. In two-way slicing, for instance,
the first halves of each partition are concatenated.
*/
template <int numBits, quantize::Type qType>
void launch_swizzled_quant_impl(int8_t* q_data,
                                float* q_scales,
                                const __half* input_data,
                                int groups,
                                int elems_per_group,
                                int pipelining,
                                int nodes,
                                int devices_per_node,
                                hipStream_t stream)
{
    debuginfo();

    // 函数首先计算一步操作中需要的线程数one_step_threads。
    // 这是基于elems_per_group和固定步长swiz_quant::h_per_step计算得出的。
    // next_pow2函数将输入值向上取到最近的2的幂。这是为了优化线程分配，因为GPU在处理2的幂次数的线程块时，效率最高。
    const int one_step_threads =
        next_pow2((elems_per_group + swiz_quant::h_per_step - 1) / swiz_quant::h_per_step);
    // 之后，它计算最大线程数max_threads，
    // 这个值是one_step_threads和预设的最大线程数swiz_quant::max_threads中的较小值。
    const int max_threads = (one_step_threads < swiz_quant::max_threads) ? one_step_threads
                                                                         : swiz_quant::max_threads;
    // 然后，它计算实际线程数threads，这个值是max_threads和预设的最小线程数swiz_quant::min_threads中的较大值。
    const int threads = (max_threads < swiz_quant::min_threads) ? swiz_quant::min_threads
                                                                : max_threads;
    // 下一步是设置CUDA的block和grid维度。block的维度是threads，
    // grid的维度则是基于分组数量，节点数和设备数计算出的。
    // 这里，每个分区的分组数groups_per_partition是总分组数groups除以总设备数
    // （节点数nodes乘以每节点设备数devices_per_node）。
    // 接着，它断言分区中的分组数可以被流水线大小pipelining整除，得到连续分组数contiguous_groups。
    // 最后，设定grid的维度，每个维度代表一个不同的并行度。
    dim3 block(threads);
    const int groups_per_partition = groups / (nodes * devices_per_node);
    assert(groups_per_partition % pipelining == 0);
    const int contiguous_groups = groups_per_partition / pipelining;
    const int partitions = nodes * devices_per_node;
    dim3 grid(contiguous_groups, pipelining, partitions);

    // elems_per_step和total_unroll是关于处理步长和展开程度的参数，它们影响kernel的并行性和计算复杂度。
    const int elems_per_step = threads * swiz_quant::h_per_step;
    const int external_unroll = ((elems_per_group + elems_per_step - 1) / elems_per_step);
    const int total_unroll = external_unroll * swiz_quant::step_granularity;

    // 接下来的一系列判断和宏调用LAUNCH_SWIZZLE_QUANT，就是基于不同的线程数和展开程度，
    // 选择并启动相应的量化kernel。不同的量化kernel在执行效率和处理数据规模方面有各自的优化。
    assert(total_unroll % 2 == 0);

    if (threads == 32) {
        LAUNCH_SWIZZLE_QUANT(2, 32);
    } else if (threads == 64) {
        LAUNCH_SWIZZLE_QUANT(2, 64);
    } else if (threads == 128) {
        LAUNCH_SWIZZLE_QUANT(2, 128);
    } else if (threads == 256) {
        LAUNCH_SWIZZLE_QUANT(2, 256);
    } else if (threads == 512) {
        if (total_unroll == 2) {
            LAUNCH_SWIZZLE_QUANT(2, 512);
        } else if (total_unroll == 4) {
            LAUNCH_SWIZZLE_QUANT(4, 512);
        } else if (total_unroll == 6) {
            LAUNCH_SWIZZLE_QUANT(6, 512);
        } else if (total_unroll == 8) {
            LAUNCH_SWIZZLE_QUANT(8, 512);
        } else if (total_unroll == 10) {
            LAUNCH_SWIZZLE_QUANT(10, 512);
        }
    }
}

// DISPATCH_SWIZZLE_QUANT宏接收两个参数num_bits和qtype，并调用了一个模板函数launch_swizzled_quant_impl，
// 这个模板函数的模板参数为num_bits和qtype，函数参数为一系列传入的值。
#define DISPATCH_SWIZZLE_QUANT(num_bits, qtype)                   \
    launch_swizzled_quant_impl<num_bits, qtype>(q_data,           \
                                                q_scales,         \
                                                input_data,       \
                                                groups,           \
                                                elems_per_group,  \
                                                pipelining,       \
                                                nodes,            \
                                                devices_per_node, \
                                                stream);
// 这个函数主要是用来根据量化的位数num_bits和量化类型q_type来调用相应的模板函数。
// 函数的参数列表包含了数据指针q_data, q_scales和input_data，这些都是在GPU内存上的数据。
// 其它的参数如groups, elems_per_group, pipelining, nodes,
// devices_per_node, stream都是用来控制量化操作的参数
void launch_swizzled_quant(int8_t* q_data,
                           float* q_scales,
                           const __half* input_data,
                           int num_bits,
                           quantize::Type q_type,
                           int groups,
                           int elems_per_group,
                           int pipelining,
                           int nodes,
                           int devices_per_node,
                           hipStream_t stream)
{
    debuginfo();

    // 如果num_bits等于4，那么就会进入第一个if分支；如果num_bits等于8，就会进入第二个if分支。
    // 在每个if分支中，都会再根据q_type的值来调用不同的模板函数。
    if (num_bits == 4) {
     // 如果q_type等于quantize::Type::Asymmetric，那么就会调用launch_swizzled_quant_impl
     // 模板函数并将模板参数设置为4和quantize::Type::Asymmetric
        if (q_type == quantize::Type::Asymmetric) {
            DISPATCH_SWIZZLE_QUANT(4, quantize::Type::Asymmetric);
        }
    // 如果q_type等于quantize::Type::Symmetric，那么就会调用launch_swizzled_quant_impl
    // 模板函数并将模板参数设置为4和quantize::Type::Symmetric。
    else if (q_type == quantize::Type::Symmetric) {
            DISPATCH_SWIZZLE_QUANT(4, quantize::Type::Symmetric);
        }
    } else if (num_bits == 8) {
        if (q_type == quantize::Type::Asymmetric) {
            DISPATCH_SWIZZLE_QUANT(8, quantize::Type::Asymmetric);
        } else if (q_type == quantize::Type::Symmetric) {
            DISPATCH_SWIZZLE_QUANT(8, quantize::Type::Symmetric);
        }
    }
}
