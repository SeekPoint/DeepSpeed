#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// SPDX-License-Identifier: Apache-2.0

// DeepSpeed Team

#include <cstdio>
#include "dequantization_utils.h"
#include "ds_kernel_utils.h"
#include "memory_access_utils.h"
#include "quantization_utils.h"
#include "reduction_utils.h"

using rop = reduce::ROpType;

/*
TODO(cmikeh2): Add implementation that better handles larger nodes. It would like make sense
to leverage some parallel reductions here to improve performance.
*/
// 这段 CUDA kernel 是用于将一些输入数据进行反量化和归约操作的。它的功能是将输入的量化数据（int8类型）
// 转换回浮点数据（__half2类型，也就是半精度浮点数），然后进行一些归约操作，并再次量化数据并输出。
// 这是一个模板函数，可以通过模板参数调整数据位宽（numBits）、张量数量（numTensors）
// 、需要处理的数据块的数量（totalChunks）、以及量化类型（quantType）：
template <int numBits, int numTensors, int totalChunks, quantize::Type quantType>
// 该 CUDA kernel 配置了一些输入和输出参数，包括输入和输出的数据和缩放因子、每个输出组的元素数量、
// 每个输入张量的元素数量、每个输入张量的组数量、每个输入组的元素数量，以及张量的总数：
__global__ void __launch_bounds__(1024) dequant_reduce(int8_t* reduced_data,
                                                       float* reduced_scales,
                                                       const int8_t* input_data,
                                                       const float* input_scales,
                                                       int elems_per_out_group,
                                                       int elems_per_in_tensor,
                                                       int groups_per_in_tensor,
                                                       int elems_per_in_group,
                                                       int num_tensors)
{
    // 这段代码首先获取了当前的线程块（tb）和线程块内的一个 warp（warp）：
    cg::thread_block tb = cg::this_thread_block();
    cg::thread_block_tile<hw_warp_size> warp = cg::tiled_partition<hw_warp_size>(tb);

    // NOTE(cmikeh2): This probably could be hardcoded to a larger number,
    // but that means even stronger restrictions on the number of elements per group
    // A performance analysis here might be beneficial
    // 根据模板参数 numBits，这段代码确定了每次内存加载的元素数量（elems_per_load）和用于存储的值的数量（storage_values）：
    constexpr int mem_granularity = (numBits == 8) ? 8 : 4;
    constexpr int elems_per_load = mem_granularity / sizeof(int8_t);  // div by 1
    constexpr int storage_values = 16 / sizeof(__half2);

    // 然后，这段代码计算了每个线程块和每个线程的偏移量，以及每次迭代的步长
    const int block_offset = tb.group_index().x * elems_per_out_group;
    const int elem_offset = tb.thread_index().x * elems_per_load;
    const int base_offset = block_offset + elem_offset;
    const int stride = tb.group_dim().x * elems_per_load;

    // 接下来，这段代码为每个线程分配了一个本地缓冲区，并初始化了一个统计对象：
    __half2 local_buffer[totalChunks * storage_values];

    quantize::GroupStats<quantType> stats;

    // 这段代码是在一个更大的循环中，其中 i 是从 0 到 totalChunks 的索引。
    // 这个循环处理的每一个“块”都包含了 storage_values 的元素。
    // #pragma unroll 是一个编译器指令，意思是编译器应该将循环展开，以减少循环的开销。
#pragma unroll
    for (int i = 0; i < totalChunks; i++) {
        // 在每个块中，首先获取一个指向当前块在 local_buffer 中开始位置的指针 iteration_buffer
        __half2* iteration_buffer = local_buffer + i * storage_values;

        // 然后，初始化 iteration_buffer 的每一个元素。reduce::init<rop::Add, __half2>()
        // 是一个模板函数，根据给定的类型和运算，返回相应的初始值。这里，初始值是加法操作的中性元素，对于加法来说，就是0。
#pragma unroll
        for (int j = 0; j < storage_values; j++) {
            iteration_buffer[j] = reduce::init<rop::Add, __half2>();
        }

        // 接着，计算了一些用于后续操作的参数：
        const int iter_offset = i * stride + base_offset;
        const int iter_scale_idx = iter_offset / elems_per_in_group;
        bool do_loads = i * stride + elem_offset < elems_per_out_group;

        // 根据 numTensors 是否大于 0，执行不同的操作。如果 numTensors 大于 0，那么对每个张量执行以下操作：
        if (numTensors > 0) {
#pragma unroll
            for (int j = 0; j < numTensors; j++) {
                // 如果 do_loads 为真，从全局内存加载数据到 load_buffer；
                if (do_loads) {
                    int8_t load_buffer[elems_per_load];

                    mem_access::load_global<mem_granularity>(
                        load_buffer, input_data + j * elems_per_in_tensor + iter_offset);

                    // 创建一个参数对象 params，用于后续的反量化操作；
                    quantize::Params<quantType, numBits> params(
                        input_scales + j * groups_per_in_tensor, iter_scale_idx);

                    __half2 dequant_buffer[storage_values];
                    // 将 load_buffer 中的数据反量化，并将结果存储到 dequant_buffer；
                    dequantize::chunk<numBits, quantType>(dequant_buffer, load_buffer, params);

#pragma unroll
                    // 将 dequant_buffer 中的每个元素添加到 iteration_buffer 对应的元素。
                    // 这里的 #pragma unroll 指令又告诉编译器将内部的循环展开。
                    for (int k = 0; k < storage_values; k++) {
                        iteration_buffer[k] =
                            reduce::element<rop::Add>(iteration_buffer[k], dequant_buffer[k]);
                    }
                }
            }
        } else {
            // 如果 numTensors 不大于 0，那么对 num_tensors 个张量执行类似的操作。这里的 #pragma unroll 4 指令告诉编译器将内部的循环展开4次。
#pragma unroll 4
            for (int j = 0; j < num_tensors; j++) {
                if (do_loads) {
                    int8_t load_buffer[elems_per_load];

                    mem_access::load_global<mem_granularity>(
                        load_buffer, input_data + j * elems_per_in_tensor + iter_offset);

                    quantize::Params<quantType, numBits> params(
                        input_scales + j * groups_per_in_tensor, iter_scale_idx);

                    __half2 dequant_buffer[storage_values];
                    dequantize::chunk<numBits, quantType>(dequant_buffer, load_buffer, params);

#pragma unroll
                    for (int k = 0; k < storage_values; k++) {
                        iteration_buffer[k] =
                            reduce::element<rop::Add>(iteration_buffer[k], dequant_buffer[k]);
                    }
                }
            }
        }
        // 最后，将 iteration_buffer 中的每个元素更新到 stats 对象。
#pragma unroll
        for (int j = 0; j < storage_values; j++) { stats.update(iteration_buffer[j]); }
    }

    // stats 是一个 quantize::GroupStats<quantType> 类型的对象，其中 quantType 是模板参数。
    // get_params 是这个类的成员函数，接收两个参数，分别是当前线程块 tb 和 warp warp，
    // 并且有两个模板参数 numBits 和 threads_per_group(1024)。
    // 这个函数的返回值是一种参数类型的对象，具体的类型取决于 quantize::GroupStats<quantType> 的定义。
    auto params = stats.template get_params<numBits, 1024>(tb, warp);

    // 然后，如果当前线程是线程块的第一个线程，那么将参数存储到 reduced_scales 中，索引是线程块的索引：
    if (tb.thread_index().x == 0) { params.store(reduced_scales, tb.group_index().x); }

    // 接下来，这段代码再次进行多次循环，每次处理一个数据块。在每个数据块内，如果条件满足，
    // 那么将本地缓冲区的数据进行量化操作，并将结果存储到输出数据：
#pragma unroll
    for (int i = 0; i < totalChunks; i++) {
        const int iter_offset = i * stride + base_offset;
        if (i * stride + elem_offset < elems_per_out_group) {
            int8_t local_output[elems_per_load];
            // 这里的 quantize::_chunk 是一个模板函数，接收三个参数，分别是存储位置 local_output、
            // 输入数据 local_buffer + i * storage_values 和参数 params，
            // 并且有两个模板参数 numBits 和 quantType。这个函数的功能是将输入数据进行量化操作，并将结果存储到
            // local_output。之后，mem_access::store_global 将 local_output 中的数据存储到 reduced_data + iter_offset。
            quantize::_chunk<numBits, quantType>(
                local_output, local_buffer + i * storage_values, params);
            mem_access::store_global<mem_granularity>(reduced_data + iter_offset, local_output);
        }
    }
}

// 这是一个C++模板函数，名称为pow2_round，它接受一个模板参数Power，并有一个整型参数raw_value。
// 这个函数的功能是将输入的raw_value向上取到最接近的2的Power次方的倍数。
// 如果Power为2（即，我们希望结果是4的倍数），且raw_value为6，那么这个函数会返回8，因为8是最接近6的4的倍数。
template <int Power>
// int32_t pow2_round(int32_t raw_value)：这是函数的定义，函数名为pow2_round，
// 返回类型为int32_t，并接受一个类型为int32_t的参数raw_value。
int32_t pow2_round(int32_t raw_value)
{
    // 首先，raw_value - 1是将raw_value减1，然后>> Power是将结果右移Power位，
    // 这相当于除以2^Power。之后，+ 1是将结果加1，这实现了向上取整。
    // 最后，<< Power是将结果左移Power位，这相当于乘以2^Power，这样就得到了最接近的2的Power次方的倍数。
    return (((raw_value - 1) >> Power) + 1) << Power;
}

#define LAUNCH_DEQUANT_REDUCE(num_chunks)                      \
    dequant_reduce<numBits, numTensors, num_chunks, quantType> \
        <<<grid, block, 0, stream>>>(reduced_data,             \
                                     reduced_scales,           \
                                     input_data,               \
                                     input_scales,             \
                                     elems_per_out_group,      \
                                     elems_per_in_tensor,      \
                                     groups_per_in_tensor,     \
                                     elems_per_in_group,       \
                                     num_tensors);

// 这个C++模板函数 launch_dequant_reduce_impl 是用于启动反量化和数据规约的CUDA kernel。
// 该函数包含三个模板参数，numBits，numTensors和quantType，这些参数在编译时必须被确定。
template <int numBits, int numTensors, quantize::Type quantType>
void launch_dequant_reduce_impl(int8_t* reduced_data,
                                float* reduced_scales,
                                const int8_t* input_data,
                                const float* input_scales,
                                int out_groups,
                                int elems_per_out_group,
                                int elems_per_in_tensor,
                                int groups_per_in_tensor,
                                int elems_per_in_group,
                                int num_tensors,
                                hipStream_t stream)
{
    // This is a coincidence. This is derived by 8 halves per 16 bytes with 2-way packing for int4
    // 定义了每个线程需要处理的元素数量，这个值与numBits（模板参数）相同。
    constexpr int elems_per_thread = numBits;
    // 计算处理一组输出元素需要的线程数，这个值取决于每个线程处理的元素数量和每个输出组的元素数量。
    // next_pow2函数计算最接近且大于等于其参数的2的幂。
    const int one_step_threads =
        next_pow2((elems_per_out_group + elems_per_thread - 1) / (elems_per_thread));
    // TODO(cmikeh2): Tune this
    // 确定线程数，如果一步所需的线程数小于1024，则使用这个值，否则使用1024。
    const int threads = (one_step_threads < 1024) ? one_step_threads : 1024;

    // 设置CUDA网格和块的维度。每个块中有threads个线程，而网格中有out_groups个块。
    dim3 block(threads);
    dim3 grid(out_groups);

    // 计算每步要处理的元素数量，这取决于线程数和每个线程处理的元素数。
    const int elems_per_step = threads * elems_per_thread;
    // 计算unroll需要多少步，取决于每个输出组中的元素数量和每一步要处理的元素数量
    const int unroll_raw = (elems_per_out_group + elems_per_step - 1) / elems_per_step;

    // 如果原始值大于等于4，那么就用2的幂进行近似，否则保持不变。
    const int unroll = (unroll_raw >= 4) ? pow2_round<1>(unroll_raw) : unroll_raw;

    // 根据优化后的unroll，调用不同的反量化和数据规约kernel。
    if (unroll == 1) {
        // 0-4096 elems
        LAUNCH_DEQUANT_REDUCE(1);
    } else if (unroll == 2) {
        // 4097-8192 etc...
        LAUNCH_DEQUANT_REDUCE(2);
    } else if (unroll == 3) {
        LAUNCH_DEQUANT_REDUCE(3);
    } else if (unroll == 4) {
        LAUNCH_DEQUANT_REDUCE(4);
    } else if (unroll == 6) {
        LAUNCH_DEQUANT_REDUCE(6);
    } else if (unroll == 8) {
        LAUNCH_DEQUANT_REDUCE(8);
    } else if (unroll == 10) {
        LAUNCH_DEQUANT_REDUCE(10);
    } else if (unroll == 12) {
        // 48k limit
        LAUNCH_DEQUANT_REDUCE(12);
    } else {
        assert(false);
    }
}

// 这是一个C++预处理器宏定义。预处理器宏是在编译时，即在源代码被转换为机器语言之前进行替换的一种机制。
// 在这个宏定义中，LAUNCH_DEQUANT_REDUCE_IMPL是宏名称，而NUM_BITS，NUM_GPUS和QUANT_TYPE是宏参数。
// 宏定义的主体是一个函数调用 launch_dequant_reduce_impl<NUM_BITS, NUM_GPUS, QUANT_TYPE>。
// 该函数是模板函数，NUM_BITS，NUM_GPUS和QUANT_TYPE是模板参数。
// 当这个宏在源代码中被使用时，例如LAUNCH_DEQUANT_REDUCE_IMPL(4, 8, quantize::Type::Symmetric)，
// 预处理器会将这个宏调用替换为launch_dequant_reduce_impl<4, 8, quantize::Type::Symmetric>
// 的函数调用，并将后面的参数列表插入到这个函数调用中。
#define LAUNCH_DEQUANT_REDUCE_IMPL(NUM_BITS, NUM_GPUS, QUANT_TYPE)                   \
    launch_dequant_reduce_impl<NUM_BITS, NUM_GPUS, QUANT_TYPE>(reduced_data,         \
                                                               reduced_scales,       \
                                                               input_data,           \
                                                               input_scales,         \
                                                               out_groups,           \
                                                               elems_per_out_group,  \
                                                               elems_per_in_tensor,  \
                                                               groups_per_in_tensor, \
                                                               elems_per_in_group,   \
                                                               num_gpus,             \
                                                               stream);
// 这个函数的作用是处理量化后的数据，将它们"反量化"并进行规约。
void launch_dequant_reduce(int8_t* reduced_data, //这是一个指针，指向存储反量化和reduce操作后的结果的内存位置。
                           float* reduced_scales, //这是一个指针，指向存储缩放因子的内存位置，这些缩放因子应用于反量化操作。
                           const int8_t* input_data, // 这是一个指向输入数据（已经量化）的常量指针。
                           const float* input_scales, // 这是一个指向输入数据量化时使用的缩放因子的常量指针。
                           int num_gpus, // 指示执行此操作的GPU数量。
                           int num_bits, // 指示用于量化操作的位数（4或8位）。
                           quantize::Type quant_type, // 指定了量化操作的类型（对称或非对称）。
                           int out_groups, // 这些是与输入数据和输出数据的维度或组相关的参数。
                           int elems_per_out_group,
                           int elems_per_in_tensor,
                           int groups_per_in_tensor,
                           int elems_per_in_group,
                           hipStream_t stream)
{
    // 根据量化类型（对称或非对称）和位数（4或8），对应的反量化和reduce的实现(LAUNCH_DEQUANT_REDUCE_IMPL)被调用。
    // 这个实现可能会根据不同的配置优化计算过程，例如对于8个GPU和16个GPU的情况。
    if (quant_type == quantize::Type::Symmetric) {
        if (num_bits == 4) {
            if (num_gpus == 8) {
                LAUNCH_DEQUANT_REDUCE_IMPL(4, 8, quantize::Type::Symmetric);
            } else if (num_gpus == 16) {
                LAUNCH_DEQUANT_REDUCE_IMPL(4, 16, quantize::Type::Symmetric);
            } else {
                LAUNCH_DEQUANT_REDUCE_IMPL(4, -1, quantize::Type::Symmetric);
            }
        } else if (num_bits == 8) {
            if (num_gpus == 8) {
                LAUNCH_DEQUANT_REDUCE_IMPL(8, 8, quantize::Type::Symmetric);
            } else if (num_gpus == 16) {
                LAUNCH_DEQUANT_REDUCE_IMPL(8, 16, quantize::Type::Symmetric);
            } else {
                LAUNCH_DEQUANT_REDUCE_IMPL(8, -1, quantize::Type::Symmetric);
            }
        }
    } else if (quant_type == quantize::Type::Asymmetric) {
        if (num_bits == 4) {
            if (num_gpus == 8) {
                LAUNCH_DEQUANT_REDUCE_IMPL(4, 8, quantize::Type::Asymmetric);
            } else if (num_gpus == 16) {
                LAUNCH_DEQUANT_REDUCE_IMPL(4, 16, quantize::Type::Asymmetric);
            } else {
                LAUNCH_DEQUANT_REDUCE_IMPL(4, -1, quantize::Type::Asymmetric);
            }
        } else if (num_bits == 8) {
            if (num_gpus == 8) {
                LAUNCH_DEQUANT_REDUCE_IMPL(8, 8, quantize::Type::Asymmetric);
            } else if (num_gpus == 16) {
                LAUNCH_DEQUANT_REDUCE_IMPL(8, 16, quantize::Type::Asymmetric);
            } else {
                LAUNCH_DEQUANT_REDUCE_IMPL(8, -1, quantize::Type::Asymmetric);
            }
        }
    }
}